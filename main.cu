#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <SDL2/SDL.h>
#include <sys/time.h>
#include "consts.h"
#include "colors.h"
#include "sdl.h"
#include "threads.h"
#include "times.h"

void render (colorpoint *points, SDL_Renderer *renderer) {
  printf("render started\n");
  struct timeval stop, start;
  float elapsed = 0;
  int p_x = 0;
  int x, y;
  int curr = 0;
  int max_dimension = HEIGHT > WIDTH ? HEIGHT : WIDTH;
  colorpoint *cp = NULL;
  color *col = NULL, *prev_color = NULL;
  SDL_Point *pt = NULL;

  gettimeofday(&start, NULL);

  for (y = 0; y < HEIGHT; y++) {
    p_x = 0;
    for (x = 0; x < WIDTH; x++) {
      curr = y * max_dimension + x;
      cp = &points[curr];
      pt = &points[curr].p;
      col = cp->c;
      if (!col) {
        continue;
      }
      if (x == 0) {
        prev_color = col;
        continue;
      } else if (same_color(prev_color, col)) {
        continue;
      }

      render_renderer(renderer, prev_color, pt, p_x);
      prev_color = col;
      p_x = pt->x;
    }

    if (col) {
      render_renderer(renderer, col, pt, p_x);
    }
  }

  gettimeofday(&stop, NULL);

  elapsed = timedifference_msec(start, stop);
  printf("print took %f milliseconds\n", elapsed);
  SDL_RenderPresent(renderer);
}

void zoom (int x, int y, float zoom, int width, int height, rect *r, complex *z1, complex *z2) {
  // WIDTH et HEIGHT sont constantes sur tout le programme (900 et 1100)
  // zoomed_width = (900 * 0.5) / 2.0;
  /*
  int zoomed_width = (width * zoom) / 2.0;
  int zoomed_height = (height * zoom) / 2.0;
  */

  int left = (float)x - (float)x * zoom;
  int right = (float)x + (float)(width - x) * zoom;
  int top = (float)y - (float)y * zoom;
  int bottom = (float)y + (float)(height - y) * zoom;

  printf("left = %d, right = %d, top = %d, bottom = %d\n",
      left, right, top, bottom);

  /*
  int left = x > zoomed_width ? x - zoomed_width + WIDTH * ratio_x : 0;
  int right = left + zoomed_width * 2;
  int top = y > zoomed_height ? y - zoomed_height : 0;
  int bottom = top + zoomed_height * 2;
  */

  coord_to_complex2(z1, left, top, r);
  coord_to_complex2(z2, right, bottom, r);

  r->left = z1->re;
  r->top = z2->im;
  r->right = z2->re;
  r->bottom = z1->im;
}

void calculate_and_render (thread_args *args, colorpoint *points, SDL_Renderer *renderer) {
  calculate(args);
  render(points, renderer);
  printf("calculate_and_render finished\n");
}

void draw_from_user_events (thread_args *args, colorpoint *points, SDL_Renderer *renderer, rect *r) {
  complex z1, z2;
  SDL_Event event;
  int x, y;
  float zoom_ratio;

  args->iterations = MIN_ITERATIONS;

  while (SDL_WaitEvent(&event)) {
    if (event.button.type == SDL_MOUSEBUTTONDOWN) {
      printf("SDL_MOUSEBUTTONDOWN\n");
      if (SDL_GetMouseState(&x, &y) & SDL_BUTTON(SDL_BUTTON_RIGHT)) {
        printf("SDL_BUTTON_RIGHT\n");
        break;
      }
      args->iterations += (float)MIN_ITERATIONS / 5.0;
      zoom_ratio = ZOOM_RATIO;
    } else if (event.button.type == SDL_MOUSEWHEEL) {
      printf("SDL_MOUSEWHEEL\n");
      SDL_GetMouseState(&x, &y);

      args->iterations += (float)MIN_ITERATIONS / 10.0;
      if (event.wheel.y == 0) continue;

      zoom_ratio = event.wheel.y < 0 ? MOUSE_WHEEL_RATIO : 2 - MOUSE_WHEEL_RATIO;
    } else continue;

    printf("zoom_ratio = %lf\n", zoom_ratio);

    printf("Mouse state x=%d, y=%d, iter=%d\n", x, y, args->iterations);
    printf("Width %lf -> %lf / Height %lf -> %lf\n", r->left, r->right, r->bottom, r->top);

    zoom(x, y, zoom_ratio, WIDTH, HEIGHT, r, &z1, &z2);
    calculate_and_render(args, points, renderer);
  }
}

void draw (rect *r, SDL_Renderer *renderer) {
  color **p_cols;
  hipMallocManaged(&p_cols, sizeof(color *));
  *p_cols = calc_speeds(MIN_ITERATIONS);
  // on préalloue tous les points d'un seul coup
  colorpoint *points;
  hipMallocManaged(&points, HEIGHT * WIDTH * sizeof(colorpoint));
  memset(points, 0, HEIGHT * WIDTH * sizeof(colorpoint));

  thread_args *args;

  hipMallocManaged(&args, sizeof(thread_args));

  prepare_thread_args(args, r, points, p_cols);
  calculate_and_render(args, points, renderer);

  draw_from_user_events(args, points, renderer, r);

  printf("Stopped loop\n");

  hipFree(points);
  hipFree(args);
  hipFree(*p_cols);
  hipFree(p_cols);
}

int main (int argc, char **argv) {
  SDL_Window * window = NULL;
  SDL_Renderer * renderer = NULL;
  // rect r = {-0.845714, -0.261667, -0.2616671, -0.8457141};
  rect *r;
  hipMallocManaged(&r, sizeof(rect));
  r->left = -2;
  r->top = 2;
  r->bottom = -2;
  r->right = 2;

  init_sdl(&window, &renderer);

  draw(r, renderer);

  free_sdl(&window, &renderer);
  hipFree(r);
  return 0;
}
